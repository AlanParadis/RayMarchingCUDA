#include "hip/hip_runtime.h"


//compute_20,sm_20;compute_30,sm_30;compute_35,sm_35;compute_37,sm_37;compute_50,sm_50;compute_52,sm_52
#ifdef NDEBUG
#define MAIN() int CALLBACK WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, LPSTR lpCmdLine, int nCmdShow)
#endif
#ifdef _DEBUG
#define MAIN() int main()
#endif

#include <stdio.h>
#include <algorithm>
#include <iterator>
#include <set>
#include <string>

#include <GL/glew.h>
#include <GL/wglew.h>
#include <GL/freeglut.h>
#include <SFML/System.hpp>
#include <SFML/Window.hpp>
#include <SFML/Graphics.hpp>

// CUDA
#include "CUDAHelper.cuh"

#include "Settings.cuh"

#include "raymarching.cuh"

__global__ void Cuda2D(float3 *nvTabPixel, rm::RayMarching* rm)
{
	//d�terminer l'emplacement o� l'on se trouve
	int pixelX = (threadIdx.x + blockIdx.x * blockDim.x);
	int pixelY = (threadIdx.y + blockIdx.y * blockDim.y);

	//�criture du pixel
	int index = pixelX + pixelY * ImageWidth;
	//nvTabPixel[index] = make_float3(0.390625f, 0.58203125f, 0.92578125f);
	
	nvTabPixel[index] = rm->Render(pixelX, pixelY);
}

MAIN()
{
	float3* nvTabPixel;
	checkCudaErrors(hipMalloc(&nvTabPixel, ImageWidth*ImageHeight * sizeof(float3)));
	if (nvTabPixel == NULL)
	{
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	GLuint gltexture;
	GLuint pbo;
	hipGraphicsResource_t cudaPBO;

	sf::RenderWindow window(sf::VideoMode(ScreenWidth, ScreenHeight), "CUDA Ray Marching", sf::Style::Titlebar | sf::Style::Close);
	window.resetGLStates();
	window.setFramerateLimit(0);
	window.setVerticalSyncEnabled(false);

	glewInit();

	glViewport(0, 0, ScreenWidth, ScreenHeight);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0.0, ScreenWidth, ScreenHeight, 0.0, -1.0, 1.0);
	glEnable(GL_TEXTURE_2D);
	glDisable(GL_LIGHTING);
	glDisable(GL_DEPTH_TEST);

	// Unbind any textures from previous.
	glBindTexture(GL_TEXTURE_2D, 0);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

	// Create new textures.
	glGenTextures(1, &gltexture);
	glBindTexture(GL_TEXTURE_2D, gltexture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	// Create image
	//https://www.khronos.org/registry/OpenGL-Refpages/gl4/html/glTexImage2D.xhtml
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB32F, ImageWidth, ImageHeight, 0, GL_RGB, GL_FLOAT, 0);

	// Create pixel buffer boject.
	glGenBuffers(1, &pbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, ImageWidth* ImageHeight * sizeof(float3), 0, GL_STREAM_COPY);
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cudaPBO, pbo, cudaGraphicsMapFlagsNone));
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
	glBindTexture(GL_TEXTURE_2D, 0);


	dim3 blockSize(NB_THREAD, 1, 1);
	dim3 gridSize(ImageWidth / (blockSize.x), (ImageHeight / blockSize.y));
	int sharedMemSize = blockSize.x * sizeof(int);
    

	rm::RayMarching* raymarching;

	checkCudaErrors(hipMallocManaged(&raymarching, sizeof(rm::RayMarching)));

	if (raymarching == nullptr)
	{
		std::cerr << "Cannot allocate memory for the ray marching engine" << std::endl;
		return -1;
	}

    raymarching->Init(&window);
    
	sf::Clock deltaClock;
	int compteur = 0;

	while (window.isOpen())
	{
		sf::Event event;
		while (window.pollEvent(event))
		{
			if (event.type == sf::Event::Closed)
			{
				exit(0);
			}
			if (event.type == sf::Event::KeyPressed)
			{
				if (event.key.code == sf::Keyboard::Escape) exit(0);
			}
            raymarching->Event(&window, &event);
		}
        
		raymarching->Update(&window, deltaClock.restart().asSeconds());

		checkCudaErrors(hipGraphicsMapResources(1, &cudaPBO, 0));
		size_t numBytes;
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&nvTabPixel, &numBytes, cudaPBO));

		//Cuda2D <<< gridSize, blockSize, sharedMemSize >>> (nvTabPixel);
		Cuda2D CUDA_KERNEL(gridSize, blockSize, sharedMemSize)(nvTabPixel, raymarching);
        
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipGraphicsUnmapResources(1, &cudaPBO, 0));


		glColor3f(1.0f, 1.0f, 1.0f);

		glBindTexture(GL_TEXTURE_2D, gltexture);
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
		glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, ImageWidth, ImageHeight, GL_RGB, GL_FLOAT, 0);

		glBegin(GL_QUADS);
		glTexCoord2f(0.0f, 0.0f);
		glVertex2f(0.0f, float(ScreenHeight));
		glTexCoord2f(1.0f, 0.0f);
		glVertex2f(float(ScreenWidth), float(ScreenHeight));
		glTexCoord2f(1.0f, 1.0f);
		glVertex2f(float(ScreenWidth), 0.0f);
		glTexCoord2f(0.0f, 1.0f);
		glVertex2f(0.0f, 0.0f);
		glEnd();

		glFlush();

		// Release
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
		glBindTexture(GL_TEXTURE_2D, 0);

		window.display();
		//	exit(0);
	}

	raymarching->Shutdown();

	hipFree(raymarching);

	return 0;
}

