#include "hip/hip_runtime.h"
#include "raymarching.cuh"

#include <math.h>

#include "Settings.cuh"
#include"SignedDistanceFunctions.cuh"
#include "Utils.cuh"

using namespace rm;

#pragma region Utils

__device__ __host__
inline float3 rm::RotatePoint(float3 point, mat3 rotation, float3 origin)
{
    return rotation * (point - origin) + origin;
}

#pragma endregion

// Signed Distance Function for the world
__device__
float RayMarching::MapTheWorld(float3 _p)
{
    // create 2 mandelbuld next to each other by 1 unit and rotate them
    float3 p1 = RotatePoint(_p, mat3::rotateY(-time), make_float3(0.0f, 0.0f, 0.0f));
    float3 p2 = RotatePoint(_p + make_float3(1.0f,0.0f,0.0f), mat3::rotateY(time), make_float3(0.0f, 0.0f, 0.0f));
    return SmoothMin(Mandelbulb(p1), Mandelbulb(p2), 0.01f);

    return MengerCube(_p);
    
    float3 sphere_0Pos = make_float3(0.0f, 0.0f, 2.0f);
    float sphere_0 = DistanceFromSphere(_p, sphere_0Pos, 0.5f);
    //float sphere_1 = DistanceFromSphere(_p, make_float3(-0.5f,0.75f,2.0f), 0.5f);

    float3 pos = rm::RotatePoint(make_float3(-0.5f, 0.4f, 2.0f), mat3::rotateY(time), sphere_0Pos);
    float sphere_1 = DistanceFromSphere(_p, pos, 0.2f);
    
    pos = rm::RotatePoint(make_float3(-0.5f, 0.6f, 2.0f), mat3::rotateY(time*2.0f), sphere_0Pos);
    float sphere_2 = DistanceFromSphere(_p, pos, 0.2f);
    
    float plane_0 = DistanceFromPlane(_p, -0.25f);

    // return min union
    return SmoothMin(SmoothMin(SmoothMin(sphere_0, sphere_1, 0.25f), sphere_2, 0.5f),plane_0, 1.0f);
}

// Apply Beer-Lambert law to generate distance based fog
__device__
float3 RayMarching::ApplyBeerLambert(float3 color, float distanceTraveled, float absorptionCoefficient)
{
    // calculate light absorption using Beer-Lambert law
    float absorption = exp(-absorptionCoefficient * distanceTraveled);
    
    return color * absorption + FOG_COLOR * (1.f - absorption);
}

__device__
float3 RayMarching::gradient(float t)
{
    float3 red = make_float3(1.f, 0.f, 0.f);
    float3 green = make_float3(0.f, 1.f, 0.f);
    float3 blue = make_float3(0.f, 0.f, 1.f);

    if (t < 1.0f / 3.0f)
    {
        // Interpolate from red to green
        return lerp(red, green, t * 3.0f);
    }
    else if (t < 2.0f / 3.0f)
    {
        // Interpolate from green to blue
        return lerp(green, blue, (t - 1.0f / 3.0f) * 3.0f);
    }
    else
    {
        // Interpolate from blue to red
        return lerp(blue, red, (t - 2.0f / 3.0f) * 3.0f);
    }
}

__device__
float3 RayMarching::ColorFromOrbitTrap(float3 currentPosition, float3 orbitTrap)
{
    // Calculate the distance from the current position to the orbit trap
    float distance = length(currentPosition - orbitTrap);

    // Use the distance to generate a color value
    // Calculate the blend factor between the current color and the next color
    float blendFactor = (1.0f + sin((distance) * 3.1415f)) / 2.0f;
    
    // Blend between the current color and the next color
    return gradient(blendFactor);
}

// from: https://jamie-wong.com/2016/07/15/ray-marching-signed-distance-functions/#surface-normals-and-lighting
__device__
float3 RayMarching::CalculateNormal(float3 _p)
{
    return normalize(make_float3(
        MapTheWorld(make_float3(_p.x + EPSILON, _p.y, _p.z)) - MapTheWorld(make_float3(_p.x - EPSILON, _p.y, _p.z)),
        MapTheWorld(make_float3(_p.x, _p.y + EPSILON, _p.z)) - MapTheWorld(make_float3(_p.x, _p.y - EPSILON, _p.z)),
        MapTheWorld(make_float3(_p.x, _p.y, _p.z + EPSILON)) - MapTheWorld(make_float3(_p.x, _p.y, _p.z - EPSILON))
    ));
    
    const float3 epsilon = make_float3(EPSILON, 0.0f, 0.0f);
    
    float3 epsilon_xyy = make_float3(epsilon.x, epsilon.y, epsilon.y);
    float3 epsilon_yxy = make_float3(epsilon.y, epsilon.x, epsilon.y);
    float3 epsilon_yyx = make_float3(epsilon.y, epsilon.y, epsilon.x);
    float d0 = MapTheWorld(_p);

    float3 d1 = make_float3(
        MapTheWorld(_p - epsilon_xyy),
        MapTheWorld(_p - epsilon_yxy),
        MapTheWorld(_p - epsilon_yyx)
    );

    float3 normal = normalize(d0 - d1);

    return normalize(normal);
}

__device__
float3 RayMarching::Raymarch(float3 ro, float3 rd)
{
    float3 currentPosition = ro;
    float distanceTraveled = 0.0f;
    float distanceToClosest = 0.0f;

    for (int i = 0; i < NUMBER_OF_STEPS; ++i)
    {
        // Calculate our current position along the ray
        currentPosition = ro + rd * distanceTraveled;

        // get distance to world geometry
        distanceToClosest = MapTheWorld(currentPosition);

        // accumulate the distance traveled thus far
        distanceTraveled += distanceToClosest;
        
        if (distanceToClosest < MINIMUM_HIT_DISTANCE) // hit
        {
            // We hit something! Return red for now
            float3 normal = CalculateNormal(currentPosition);
            // For now, hard-code the light's position in our scene
            float3 lightPosition = make_float3(2.0f, -5.0f, 3.0f);
            // Calculate the unit direction vector that points from
            // the point of intersection to the light source
            float3 directionToLight = normalize(currentPosition - lightPosition);
            // Calculate the diffuse intensity
            float diffuseIntensity = max(0.0f, dot(normal, directionToLight));
            // Calculate the color of the object
            float3 baseColor = ColorFromOrbitTrap(currentPosition, make_float3(0));
            // Apply light to red colored scene
            float3 finalColor =  baseColor * diffuseIntensity;
            // Generate distance fog with Beer Lambert law
            finalColor = ApplyBeerLambert(finalColor, distanceTraveled, FOG_THICKNESS);
            
            return finalColor;
        }

        if (distanceTraveled > MAXIMUM_TRACE_DISTANCE) // miss
        {
            break;
        }
    }

    // If we get here, we didn't hit anything so just
    // return a background color
    return ApplyBeerLambert(make_float3(1.0f), distanceTraveled, FOG_THICKNESS);
}

void RayMarching::Init(sf::RenderWindow* _window)
{ 
    // camera setup
    camera.pos = make_float3(0.0f, 0.0f, 0.0f);
    camera.dir = make_float3(1.0f, 0.0f, 0.0f);
	camera.right = normalize(cross(camera.dir, make_float3(0, 1, 0)));
	camera.up = normalize(cross(camera.right, camera.dir));
	float fov = FOV / 180.0f * float(M_PI);
	camera.invhalffov = 1.0f / std::tan(fov / 2.0f);
    // grab mouse
    _window->setMouseCursorVisible(false);
    _window->setMouseCursorGrabbed(true);
    isMouseLock = true;
    // set mouse to center
    sf::Vector2i center(_window->getSize().x / 2, _window->getSize().y / 2);
    sf::Mouse::setPosition(center, *_window);

    time = 0;
}

void RayMarching::Event(sf::RenderWindow* _window, sf::Event* _evt)
{
    // middle click to toggle mouse lock
    if (_evt->type == sf::Event::MouseButtonPressed && _evt->mouseButton.button == sf::Mouse::Middle)
    {
        if (isMouseLock)
        {
            _window->setMouseCursorVisible(true);
            _window->setMouseCursorGrabbed(false);
            isMouseLock = false;
        }
        else
        {
            _window->setMouseCursorVisible(false);
            _window->setMouseCursorGrabbed(true);
            isMouseLock = true;
        }
    }
}

void RayMarching::Update(sf::RenderWindow* _window, float _dt)
{
    time += _dt;
    
    //! KEYBOARDS INPUTS
    
    // Handle user input to move the camera
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::W))
    {
        // Move camera forward
        camera.pos += camera.dir * CAM_SPEED * _dt;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::S))
    {
        // Move camera backward
        camera.pos -= camera.dir * CAM_SPEED * _dt;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::A))
    {
        // Move camera left
        camera.pos -= camera.right * CAM_SPEED * _dt;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::D))
    {
        // Move camera right
        camera.pos += camera.right * CAM_SPEED * _dt;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::E))
    {
        // Move camera up
        camera.pos += camera.up * CAM_SPEED * _dt;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::Q))
    {
        // Move camera down
        camera.pos -= camera.up * CAM_SPEED * _dt;
    }

    // no mouse look is mouse is unlocked
    if (!isMouseLock)
        return;
    
    //! MOUSE LOOK

    // Get screen center
    sf::Vector2i center(_window->getSize().x / 2.f, _window->getSize().y / 2.f);
    // get mose delta from screen center
    sf::Vector2i delta = sf::Mouse::getPosition(*_window) - center;
   
    // get rotations
    float pitch = -delta.y * CAM_SENSITIVITY;
    float yaw = delta.x * CAM_SENSITIVITY;
    
    // First, apply the yaw rotation around the camera's up vector
    camera.dir = normalize(camera.dir * cos(yaw) + cross(camera.dir, camera.up) * sin(yaw));
    // Then, apply the pitch rotation around the camera's right vector
    camera.dir = normalize(camera.dir * cos(pitch) + camera.up * sin(pitch));    
    
    // update cam right and up vector
	camera.right = normalize(cross(camera.dir, make_float3(0.f, 1.f, 0.f)));
    camera.up = normalize(cross(camera.right, camera.dir));
    
    // reset mouse in center
    sf::Mouse::setPosition(center, *_window);
}

__device__ 
float3 RayMarching::Render(int _pX, int _pY)
{
    //screen resolution
    float2 resolution = make_float2((float)ImageWidth, (float)ImageHeight);   
	// pixel coordinates
    float2 coordinates = make_float2((float)_pX, (float)_pY);
    // get screen uv
    float2 uv = ( coordinates - (resolution * 0.5f)) / resolution.y;
    // ray origin
    float3 ro = camera.pos;
    // ray direction
    float3 rd = normalize(camera.dir + uv.x * camera.right + uv.y * camera.up + camera.dir * camera.invhalffov);
    // Raymarch to find the shaded color of the fragment
    float3 shaded_color = Raymarch(ro, rd);
    // return color
    return shaded_color;
}

void RayMarching::Shutdown()
{
    
}
