#include "hip/hip_runtime.h"
#include "raymarching.cuh"

#include <math.h>

#include "Settings.cuh"
#include"SignedDistanceFunctions.cuh"

#pragma region Utils

// get sign function
#define sign(x) ((x > 0) - (x < 0))
// clamp macro
#define clamp(x, a, b) (x < a ? a : (x > b ? b : x))

__device__
inline float rm::SmoothMin(float dstA, float dstB, float k) {
    float h = std::fmaxf(k - abs(dstA - dstB), 0) / k;
    return std::fminf(dstA, dstB) - h * h * h * k * (1.0f / 6.0f);
}

#pragma endregion

__device__
float rm::RayMarching::MapTheWorld(float3 _p)
{
    float sphere_0 = DistanceFromSphere(_p, make_float3(0.0f,0.0f,2.0f), 0.5f);
    float sphere_1 = DistanceFromSphere(_p, make_float3(-0.5f,0.75f,2.0f), 0.5f);
    
    float plane_0 = DistanceFromPlane(_p, -0.25f);

    // return min union
    return SmoothMin(SmoothMin(sphere_0, sphere_1, 0.5), plane_0, 1.0);
}

// function to apply Beer-Lambert law to diffuse intensityi
__device__
float3 rm::RayMarching::ApplyBeerLambert(float3 color, float distanceTraveled, float absorptionCoefficient)
{
    // calculate light absorption using Beer-Lambert law
    float absorption = exp(-absorptionCoefficient * distanceTraveled);
    
    return color * absorption + FOG_COLOR * (1 - absorption);
}

__device__
float3 rm::RayMarching::CalculateNormal(float3 _p)
{
    return normalize(make_float3(
        MapTheWorld(make_float3(_p.x + EPSILON, _p.y, _p.z)) - MapTheWorld(make_float3(_p.x - EPSILON, _p.y, _p.z)),
        MapTheWorld(make_float3(_p.x, _p.y + EPSILON, _p.z)) - MapTheWorld(make_float3(_p.x, _p.y - EPSILON, _p.z)),
        MapTheWorld(make_float3(_p.x, _p.y, _p.z + EPSILON)) - MapTheWorld(make_float3(_p.x, _p.y, _p.z - EPSILON))
    ));
    
    const float3 epsilon = make_float3(EPSILON, 0.0f, 0.0f);
    
    float3 epsilon_xyy = make_float3(epsilon.x, epsilon.y, epsilon.y);
    float3 epsilon_yxy = make_float3(epsilon.y, epsilon.x, epsilon.y);
    float3 epsilon_yyx = make_float3(epsilon.y, epsilon.y, epsilon.x);
    float d0 = MapTheWorld(_p);

    float3 d1 = make_float3(
        MapTheWorld(_p - epsilon_xyy),
        MapTheWorld(_p - epsilon_yxy),
        MapTheWorld(_p - epsilon_yyx)
    );

    float3 normal = normalize(d0 - d1);

    return normalize(normal);
}

__device__
float3 rm::RayMarching::Raymarch(float3 ro, float3 rd)
{
    float3 currentPosition = ro;
    float distanceTraveled = 0.0f;
    float distanceToClosest = 0.0f;

    for (int i = 0; i < NUMBER_OF_STEPS; ++i)
    {
        // Calculate our current position along the ray
        currentPosition = ro + rd * distanceTraveled;

        // get distance to world geometry
        distanceToClosest = MapTheWorld(currentPosition);

        // accumulate the distance traveled thus far
        distanceTraveled += distanceToClosest;
        
        if (distanceToClosest < MINIMUM_HIT_DISTANCE) // hit
        {
            //print distance
            //break;
            
            // We hit something! Return red for now
            float3 normal = CalculateNormal(currentPosition);

            //return normal * 0.5 + 0.5;

            // For now, hard-code the light's position in our scene
            float3 lightPosition = make_float3(2.0, -5.0, 3.0);

            // Calculate the unit direction vector that points from
            // the point of intersection to the light source
            float3 directionToLight = normalize(currentPosition - lightPosition);

            float diffuseIntensity = max(0.0, dot(normal, directionToLight));

            //diffuseIntensity = //ApplyBeerLambert(diffuseIntensity, distanceTraveled, 1.5);
            float3 finalColor =  make_float3(1.0, 0.0, 0.0) * diffuseIntensity;

            finalColor = ApplyBeerLambert(finalColor, distanceTraveled, FOG_THICKNESS);
            
            return finalColor;
        }

        if (distanceTraveled > MAXIMUM_TRACE_DISTANCE) // miss
        {
            break;
        }
    }

    // If we get here, we didn't hit anything so just
    // return a background color
    return ApplyBeerLambert(make_float3(1), distanceTraveled, FOG_THICKNESS);
    //return make_float3(distanceTraveled/5);
    //return make_float3(0.390625f, 0.58203125f, 0.92578125f);
}

void rm::RayMarching::Init(sf::RenderWindow* _window)
{ 
    cam.pos = make_float3(0.0f, 0.0f, 0.0f);
    cam.dir = make_float3(0.0f, 0.0f, 1.0f);
	cam.right = normalize(cross(cam.dir, make_float3(0, 1, 0)));
	cam.up = normalize(cross(cam.right, cam.dir));
	float fov = FOV / 180.0f * float(M_PI);
	cam.invhalffov = 1.0f / std::tan(fov / 2.0f);

    //_window->setMouseCursorVisible(false);
    //_window->setMouseCursorGrabbed(true);
}

void rm::RayMarching::Update(sf::RenderWindow* _window, float _dt)
{
    /*
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::W))
    {
        cam.pos += CAM_SPEED * _dt * cam.dir;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::S))
    {
        cam.pos -= CAM_SPEED * _dt * cam.dir;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::A))
    {
        cam.pos -= CAM_SPEED * _dt * cam.right;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::D))
    {
        cam.pos += CAM_SPEED * _dt * cam.right;
    }
    */
    
}

__device__ 
float3 rm::RayMarching::Render(int _pX, int _pY)
{
    float2 resolution = make_float2((float)ImageWidth, (float)ImageHeight);   //screen resolution
	float2 coordinates = make_float2((float)_pX, (float)_pY);   //fragment coordinates
    
    //float2 uv = ( 2 * coordinates - resolution) / resolution.y;
    float2 uv = ( coordinates - (resolution * 0.5)) / resolution.y;
	
    float3 ro = make_float3(0.0f);   //ray origin
    float3 rd = normalize(make_float3(uv, 1.0f) - ro);   //ray direction
    
    float3 shaded_color = Raymarch(ro, rd);
    //float3 shaded_color = make_float3(uv.x, uv.y, 0.0f);

    return shaded_color;
}

void rm::RayMarching::Shutdown()
{
    
}
